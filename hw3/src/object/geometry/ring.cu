#include "hip/hip_runtime.h"
#include "object/geometry/ring.hpp"

using namespace px;

BaseRing::BaseRing(Point const &pos,
                   Direction const &norm_vec,
                   PREC const &radius1,
                   PREC const &radius2)
        : _center(pos), _norm(norm_vec),
          _inner_radius(radius1 < radius2 ? radius1 : radius2),
          _outer_radius(radius1 > radius2 ? radius1 : radius2),
          _inner_radius2(_inner_radius*_inner_radius),
          _outer_radius2(_outer_radius*_outer_radius),
          _p_dot_n(pos.dot(norm_vec)),
          _dev_obj(nullptr)
{}

PX_CUDA_CALLABLE
GeometryObj *BaseRing::hitCheck(void * const &obj,
                         Ray const &ray,
                         PREC const &t_start,
                         PREC const &t_end,
                         PREC &hit_at)
{
    auto o = reinterpret_cast<BaseRing*>(obj);

    auto tmp = (o->_p_dot_n - ray.original.dot(o->_norm)) / ray.direction.dot(o->_norm);
    if (tmp > t_start && tmp < t_end)
    {
        auto intersect = ray[tmp];
        auto dist2 = (intersect.x - o->_center.x) * (intersect.x - o->_center.x) +
                     (intersect.y - o->_center.y) * (intersect.y - o->_center.y) +
                     (intersect.z - o->_center.z) * (intersect.z - o->_center.z);
        if (dist2 <= o->_outer_radius2 && dist2 >= o->_inner_radius2)
        {
            hit_at = tmp;
            return o->_dev_obj;
        }
    }
    return nullptr;
}

PX_CUDA_CALLABLE
Direction BaseRing::normalVec(void * const &obj,
                              PREC const &x, PREC const &y, PREC const &z)
{
    return reinterpret_cast<BaseRing*>(obj)->_norm;
}

PX_CUDA_CALLABLE
Vec3<PREC> BaseRing::getTextureCoord(void * const &obj,
                                     PREC const &x, PREC const &y,
                                     PREC const &z)
{
    auto o = reinterpret_cast<BaseRing*>(obj);
    return {x - o->_center.x,
            o->_norm.y*(z - o->_center.z) - o->_center.z*(y - o->_center.y) ,
            (x - o->_center.x)*o->_norm.x + (y - o->_center.y)*o->_norm.y + (z - o->_center.z)*o->_norm.z};
}


void BaseRing::setCenter(Point const &center)
{
    _center = center;
    _p_dot_n = center.dot(_norm);
}

void BaseRing::setNormal(Direction const &norm_vec)
{
    _norm = norm_vec;
    _p_dot_n = _center.dot(norm_vec);
}

void BaseRing::setRadius(PREC const &radius1, PREC const &radius2)
{
    _inner_radius = std::min(radius1, radius2);
    _outer_radius = std::max(radius1, radius2);
    _inner_radius2 = _inner_radius*_inner_radius;
    _outer_radius2 = _outer_radius*_outer_radius;
}


std::shared_ptr<BaseGeometry> Ring::create(Point const &position,
                                           Direction const &norm_vec,
                                           PREC const &radius1,
                                           PREC const &radius2,
                                           std::shared_ptr<BaseMaterial> const &material,
                                           std::shared_ptr<Transformation> const &trans)
{
    return std::shared_ptr<BaseGeometry>(new Ring(position, norm_vec,
                                                  radius1, radius2,
                                                  material, trans));
}

Ring::Ring(Point const &position,
           Direction const &norm_vec,
           PREC const &radius1,
           PREC const &radius2,
           std::shared_ptr<BaseMaterial> const &material,
           std::shared_ptr<Transformation> const &trans)
        : BaseGeometry(material, trans, 8),
          _obj(new BaseRing(position, norm_vec, radius1, radius2)),
          _gpu_obj(nullptr), _need_upload(true)
{
    _obj->_dev_obj = reinterpret_cast<GeometryObj*>(this);
    _updateVertices();
}

Ring::~Ring()
{
    delete _obj;
#ifdef USE_CUDA
    clearGpuData();
#endif
}


#ifdef USE_CUDA
__device__ fnHit_t __fn_hit_ring = BaseRing::hitCheck;
__device__ fnNormal_t __fn_normal_ring = BaseRing::normalVec;
__device__ fnTextureCoord_t __fn_texture_coord_ring = BaseRing::getTextureCoord;
#endif
void Ring::up2Gpu()
{
#ifdef USE_CUDA
    static fnHit_t fn_hit_h = nullptr;
    static fnNormal_t fn_normal_h;
    static fnTextureCoord_t fn_texture_coord_h;

    if (_need_upload)
    {
        if (dev_ptr == nullptr)
        {
            PX_CUDA_CHECK(hipMalloc(&_gpu_obj, sizeof(BaseRing)));
            PX_CUDA_CHECK(hipMalloc(&dev_ptr, sizeof(GeometryObj)));
        }

        if (fn_hit_h == nullptr)
        {
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_hit_h, HIP_SYMBOL(__fn_hit_ring), sizeof(fnHit_t)));
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_normal_h, HIP_SYMBOL(__fn_normal_ring), sizeof(fnNormal_t)));
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_texture_coord_h, HIP_SYMBOL(__fn_texture_coord_ring), sizeof(fnTextureCoord_t)));
        }

        if (mat != nullptr)
            mat->up2Gpu();

        if (trans != nullptr)
            trans->up2Gpu();

        _obj->_dev_obj = dev_ptr;
        PX_CUDA_CHECK(hipMemcpy(_gpu_obj, _obj, sizeof(BaseRing), hipMemcpyHostToDevice));
        _obj->_dev_obj = reinterpret_cast<GeometryObj*>(this);

        GeometryObj tmp(_gpu_obj, fn_hit_h, fn_normal_h, fn_texture_coord_h,
                        mat == nullptr ? nullptr : mat->devPtr(),
                        trans == nullptr ? nullptr : trans->devPtr());

        PX_CUDA_CHECK(hipMemcpy(dev_ptr, &tmp, sizeof(GeometryObj),
                                 hipMemcpyHostToDevice))

        _need_upload = false;
    }
#endif
}

void Ring::clearGpuData()
{
#ifdef USE_CUDA
    BaseGeometry::clearGpuData();
    if (_gpu_obj != nullptr)
    {
        PX_CUDA_CHECK(hipFree(_gpu_obj));
        _gpu_obj = nullptr;
    }
    _need_upload = true;
#endif
}


void Ring::setCenter(Point const &center)
{
    _obj->setCenter(center);
    _updateVertices();
#ifdef USE_CUDA
    _need_upload = true;
#endif
}

void Ring::setNormal(Direction const &norm)
{
    _obj->setNormal(norm);
    _updateVertices();
#ifdef USE_CUDA
    _need_upload = true;
#endif
}

void Ring::setRadius(PREC const &radius1, PREC const &radius2)
{
    _obj->setRadius(radius1, radius2);
    _updateVertices();
#ifdef USE_CUDA
    _need_upload = true;
#endif
}

void Ring::_updateVertices()
{
    raw_vertices[0].x = _obj->_center.x + _obj->_outer_radius;
    raw_vertices[0].y = _obj->_center.y + _obj->_outer_radius;
    raw_vertices[0].z = _obj->_center.z + _obj->_outer_radius;

    raw_vertices[1].x = _obj->_center.x - _obj->_outer_radius;
    raw_vertices[1].y = _obj->_center.y + _obj->_outer_radius;
    raw_vertices[1].z = _obj->_center.z + _obj->_outer_radius;

    raw_vertices[2].x = _obj->_center.x + _obj->_outer_radius;
    raw_vertices[2].y = _obj->_center.y - _obj->_outer_radius;
    raw_vertices[2].z = _obj->_center.z + _obj->_outer_radius;

    raw_vertices[3].x = _obj->_center.x + _obj->_outer_radius;
    raw_vertices[3].y = _obj->_center.y + _obj->_outer_radius;
    raw_vertices[3].z = _obj->_center.z - _obj->_outer_radius;

    raw_vertices[4].x = _obj->_center.x - _obj->_outer_radius;
    raw_vertices[4].y = _obj->_center.y - _obj->_outer_radius;
    raw_vertices[4].z = _obj->_center.z + _obj->_outer_radius;

    raw_vertices[5].x = _obj->_center.x - _obj->_outer_radius;
    raw_vertices[5].y = _obj->_center.y + _obj->_outer_radius;
    raw_vertices[5].z = _obj->_center.z - _obj->_outer_radius;

    raw_vertices[6].x = _obj->_center.x + _obj->_outer_radius;
    raw_vertices[6].y = _obj->_center.y - _obj->_outer_radius;
    raw_vertices[6].z = _obj->_center.z - _obj->_outer_radius;

    raw_vertices[7].x = _obj->_center.x - _obj->_outer_radius;
    raw_vertices[7].y = _obj->_center.y - _obj->_outer_radius;
    raw_vertices[7].z = _obj->_center.z - _obj->_outer_radius;
}

Vec3<PREC> Ring::getTextureCoord(PREC const &x,
                                 PREC const &y,
                                 PREC const &z) const
{
    return BaseRing::getTextureCoord(_obj, x, y, z);
}
const BaseGeometry *Ring::hitCheck(Ray const &ray,
                                   PREC const &t_start,
                                   PREC const &t_end,
                                   PREC &hit_at) const
{
    return BaseRing::hitCheck(_obj, ray, t_start, t_end, hit_at) ? this : nullptr;
}
Direction Ring::normalVec(PREC const &x, PREC const &y,
                          PREC const &z) const
{
    return BaseRing::normalVec(_obj, x, y, z);
}

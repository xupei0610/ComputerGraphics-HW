#include "hip/hip_runtime.h"
#include "object/geometry/ring.hpp"

#ifdef USE_CUDA
#include "gpu_creator.hpp"
#endif

using namespace px;

PX_CUDA_CALLABLE
BaseRing::BaseRing(Point const &pos,
                   Direction const &norm_vec,
                   PREC const &radius1,
                   PREC const &radius2,
                   const BaseMaterial *const &material,
                   const Transformation *const &trans)
        : BaseGeometry(material, trans, 8),
          _center(pos),
          _inner_radius(radius1 < radius2 ? radius1 : radius2),
          _outer_radius(radius1 > radius2 ? radius1 : radius2),
          _inner_radius2(_inner_radius*_inner_radius),
          _outer_radius2(_outer_radius*_outer_radius),
          _p_dot_n(pos.dot(norm_vec))
{
    updateVertices();
}

PX_CUDA_CALLABLE
const BaseGeometry * BaseRing::hitCheck(Ray const &ray,
                                  PREC const &t_start,
                                  PREC const &t_end,
                                  PREC &hit_at) const
{
    auto tmp = (_p_dot_n - ray.original.dot(_norm_vec)) / ray.direction.dot(_norm_vec);
    if (tmp > t_start && tmp < t_end)
    {
        auto intersect = ray[tmp];
        auto dist2 = (intersect.x - _center.x) * (intersect.x - _center.x) +
                     (intersect.y - _center.y) * (intersect.y - _center.y) +
                     (intersect.z - _center.z) * (intersect.z - _center.z);
        if (dist2 <= _outer_radius2 && dist2 >= _inner_radius2)
        {
            hit_at = tmp;
            return this;
        }
    }
    return nullptr;
}

PX_CUDA_CALLABLE
Direction BaseRing::normalVec(PREC const &x, PREC const &y, PREC const &z) const
{
    return _norm_vec;
}

PX_CUDA_CALLABLE
Vec3<PREC> BaseRing::getTextureCoord(PREC const &x, PREC const &y,
                                       PREC const &z) const
{
    return {x - _center.x,
            -_norm_vec.z*(y - _center.y) + _norm_vec.y*(z - _center.z),
            (x - _center.x)*_norm_vec.x + (y - _center.y)*_norm_vec.y + (z - _center.z)*_norm_vec.z};
}

std::shared_ptr<Geometry> Ring::create(Point const &position,
                                           Direction const &norm_vec,
                                           PREC const &radius1,
                                           PREC const &radius2,
                                           std::shared_ptr<Material> const &material,
                                           std::shared_ptr<Transformation> const &trans)
{
    return std::shared_ptr<Geometry>(new Ring(position, norm_vec,
                                                  radius1, radius2,
                                                  material, trans));
}

Ring::Ring(Point const &position,
           Direction const &norm_vec,
           PREC const &radius1,
           PREC const &radius2,
           std::shared_ptr<Material> const &material,
           std::shared_ptr<Transformation> const &trans)
        : _obj(new BaseRing(position, norm_vec, radius1, radius2, material->obj(), trans.get())),
          _base_obj(_obj),
          _material_ptr(material), _transformation_ptr(trans),
          _dev_ptr(nullptr), _need_upload(true)
{}

Ring::~Ring()
{
    delete _obj;
#ifdef USE_CUDA
    clearGpuData();
#endif
}

BaseGeometry * const &Ring::obj() const noexcept
{
    return _base_obj;
}

BaseGeometry **Ring::devPtr()
{
    return _dev_ptr;
}

void Ring::up2Gpu()
{
#ifdef USE_CUDA
    if (_need_upload)
    {
        clearGpuData();
        PX_CUDA_CHECK(hipMalloc(&_dev_ptr, sizeof(BaseGeometry **)));

        if (_material_ptr != nullptr)
            _material_ptr->up2Gpu();
        if (_transformation_ptr != nullptr)
            _transformation_ptr->up2Gpu();

        hipDeviceSynchronize();

        GpuCreator::Ring(_dev_ptr,
                         _obj->_center, _obj->_norm_vec, _obj->_inner_radius, _obj->_outer_radius,
                        _material_ptr == nullptr ? nullptr : _material_ptr->devPtr(),
                        _transformation_ptr == nullptr ? nullptr : _transformation_ptr->devPtr());

        _need_upload = false;
    }
#endif
}

void Ring::clearGpuData()
{
#ifdef USE_CUDA
    if (_dev_ptr == nullptr)
        return;

    if (_transformation_ptr.use_count() == 1)
        _transformation_ptr->clearGpuData();
    if (_material_ptr.use_count() == 1)
        _material_ptr->clearGpuData();

    GpuCreator::destroy(_dev_ptr);
    _dev_ptr = nullptr;
    _need_upload = true;
#endif
}


void Ring::setCenter(Point const &center)
{
    _obj->_center = center;
    _obj->_p_dot_n = center.dot(_obj->_norm_vec);

    _obj->updateVertices();
#ifdef USE_CUDA
    _need_upload = true;
#endif
}

void Ring::setNormVec(Direction const &norm_vec)
{
    _obj->_norm_vec = norm_vec;
    _obj->_p_dot_n = _obj->_center.dot(norm_vec);

    _obj->updateVertices();
#ifdef USE_CUDA
    _need_upload = true;
#endif
}

void Ring::setRadius(PREC const &radius1, PREC const &radius2)
{
    _obj->_inner_radius = std::min(radius1, radius2);
    _obj->_outer_radius = std::max(radius1, radius2);
    _obj->_inner_radius2 = _obj->_inner_radius*_obj->_inner_radius;
    _obj->_outer_radius2 = _obj->_outer_radius*_obj->_outer_radius;

    _obj->updateVertices();
#ifdef USE_CUDA
    _need_upload = true;
#endif
}

PX_CUDA_CALLABLE
void BaseRing::updateVertices()
{
    _raw_vertices[0].x = _center.x + _outer_radius;
    _raw_vertices[0].y = _center.y + _outer_radius;
    _raw_vertices[0].z = _center.z + _outer_radius;

    _raw_vertices[1].x = _center.x - _outer_radius;
    _raw_vertices[1].y = _center.y + _outer_radius;
    _raw_vertices[1].z = _center.z + _outer_radius;

    _raw_vertices[2].x = _center.x + _outer_radius;
    _raw_vertices[2].y = _center.y - _outer_radius;
    _raw_vertices[2].z = _center.z + _outer_radius;

    _raw_vertices[3].x = _center.x + _outer_radius;
    _raw_vertices[3].y = _center.y + _outer_radius;
    _raw_vertices[3].z = _center.z - _outer_radius;

    _raw_vertices[4].x = _center.x - _outer_radius;
    _raw_vertices[4].y = _center.y - _outer_radius;
    _raw_vertices[4].z = _center.z + _outer_radius;

    _raw_vertices[5].x = _center.x - _outer_radius;
    _raw_vertices[5].y = _center.y + _outer_radius;
    _raw_vertices[5].z = _center.z - _outer_radius;

    _raw_vertices[6].x = _center.x + _outer_radius;
    _raw_vertices[6].y = _center.y - _outer_radius;
    _raw_vertices[6].z = _center.z - _outer_radius;

    _raw_vertices[7].x = _center.x - _outer_radius;
    _raw_vertices[7].y = _center.y - _outer_radius;
    _raw_vertices[7].z = _center.z - _outer_radius;
}

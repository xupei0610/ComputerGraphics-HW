#include "hip/hip_runtime.h"
#include "object/geometry/cylinder.hpp"

using namespace px;

BaseCylinder::BaseCylinder(const BaseMaterial *const &material,
                   const Transformation *const &trans)
        : BaseGeometry(material, trans, 8)
{}

PX_CUDA_CALLABLE
BaseGeometry * BaseCylinder::hitCheck(Ray const &ray,
                                  double const &t_start,
                                  double const &t_end,
                                  double &hit_at)
{
    auto xo = ray.original.x - _center.x;
    auto yo = ray.original.y - _center.y;

    // @see http://www.bmsc.washington.edu/people/merritt/graphics/quadrics.html
    auto A =  _a * ray.direction.x * ray.direction.x +
              _b * ray.direction.y * ray.direction.y;
    auto B = 2 * _a * xo * ray.direction.x +
             2 * _b * yo * ray.direction.y;
    auto C =  _a * xo * xo +
              _b * yo * yo - 1;

    bool hit_top = false;

    auto tmp1 = (_z1 - ray.original.z) / ray.direction.z;
    auto tmp_x = ray.original.x + ray.direction.x * tmp1;
    auto tmp_y = ray.original.y + ray.direction.y * tmp1;

    if (tmp1 >= t_start && tmp1 <= t_end &&
        _a * (tmp_x - _center.x) * (tmp_x - _center.x) +
        _b * (tmp_y - _center.y) * (tmp_y - _center.y) <= 1)
    {
        hit_top = true;
        hit_at = tmp1;
    }

    auto tmp2 = (_z0 - ray.original.z) / ray.direction.z;

    if (tmp1 >= t_start && tmp1 <= t_end &&
        (hit_top == false || tmp2 < tmp1))
    {
        tmp_x = ray.original.x + ray.direction.x * tmp2;
        tmp_y = ray.original.y + ray.direction.y * tmp2;

        if (_a * (tmp_x - _center.x) * (tmp_x - _center.x) +
            _b * (tmp_y - _center.y) * (tmp_y - _center.y) <= 1)
        {
            hit_top = true;
            hit_at = tmp2;
        }
    }

    if (A == 0)
    {
        if (B == 0) return nullptr;

        auto tmp = - C / B;
        if (tmp > t_start && tmp < t_end)
        {
            auto iz = ray.original.z + ray.direction.z*tmp;
            if (iz >= _z0 && iz<=_z1)
            {
                if (hit_top == false || hit_at > tmp)
                    hit_at = tmp;
                return this;
            }
        }
        return nullptr;
    }

    auto discriminant = B * B - 4 * A * C;
    if (discriminant < 0)
        return nullptr;

    discriminant = std::sqrt(discriminant);
    tmp1 = (-B - discriminant)/ (2.0 * A);
    tmp2 = (-B + discriminant)/ (2.0 * A);
    if (tmp1 > tmp2)
        std::swap(tmp1, tmp2);
    if (tmp1 > t_start && tmp1 < t_end)
    {
        auto iz = ray.original.z + ray.direction.z*tmp1;
        if (iz >= _z0 && iz<=_z1)
        {
            if (hit_top == false || hit_at > tmp1)
                hit_at = tmp1;
            return this;
        }
    }
    if (tmp2 > t_start && tmp2 < t_end)
    {
        auto iz = ray.original.z + ray.direction.z*tmp2;
        if (iz >= _z0 && iz<=_z1)
        {
            if (hit_top == false || hit_at > tmp2)
                hit_at = tmp2;
            return this;
        }
    }

    return hit_top ? this : nullptr;
}

PX_CUDA_CALLABLE
Direction BaseCylinder::normalVec(double const &x, double const &y, double const &z)
{
    if (std::abs(z - _z0) < 1e-12)
        return {0, 0, -1};
    if (std::abs(z - _z1) < 1e-12)
        return {0, 0, 1};

    return {_a * (x - _center.x),
            _b * (y - _center.y),
            0};
}

PX_CUDA_CALLABLE
Vec3<double> BaseCylinder::getTextureCoord(double const &x, double const &y,
                                       double const &z)
{
    if (std::abs(z - _z0) < 1e-12)
        return {x - _center.x,
                _radius_y + y - _center.y, 0};
    if (std::abs(z - _z1) < 1e-12)
        return {x - _center.x,
                _radius_y + _radius_y + _radius_y + _abs_height + y - _center.y, 0};

    auto dx = x - _center.x;
    auto dy = y - _center.y - _radius_y;

    return {((_a/3.0 * dx * dx * dx - dx) + _b/3.0 * dy * dy * dy),
            _radius_y + _radius_y + z - _center.z, 0};
}

std::shared_ptr<BaseGeometry> Cylinder::create(Point const &center_of_bottom_face,
                                               double const &radius_x, double const &radius_y,
                                               double const &height,
                                           std::shared_ptr<BaseMaterial> const &material,
                                           std::shared_ptr<Transformation> const &trans)
{
    return std::shared_ptr<BaseGeometry>(new Cylinder(center_of_bottom_face,
                                                      radius_x, radius_y,
                                                      height,
                                                      material, trans));
}

Cylinder::Cylinder(Point const &center_of_bottom_face,
                   double const &radius_x, double const &radius_y,
                   double const &height,
           std::shared_ptr<BaseMaterial> const &material,
           std::shared_ptr<Transformation> const &trans)
        : BaseCylinder(material.get(), trans.get()),
          _material_ptr(material), _transformation_ptr(trans),
          _dev_ptr(nullptr), _need_upload(true)
{
    setParams(center_of_bottom_face,
              radius_x, radius_y,
              height);
}

Cylinder::~Cylinder()
{
#ifdef USE_CUDA
    clearGpuData();
#endif
}

BaseGeometry *Cylinder::up2Gpu()
{
#ifdef USE_CUDA
    if (_need_upload)
    {
        if (_dev_ptr == nullptr)
            PX_CUDA_CHECK(hipMalloc(&_dev_ptr, sizeof(BaseCylinder)));

        material = _material_ptr->up2Gpu();
        transformation = _transformation_ptr->up2Gpu();

        PX_CUDA_CHECK(hipMemcpy(_dev_ptr,
                                 dynamic_cast<BaseCylinder*>(this),
                                 sizeof(BaseCylinder),
                                 hipMemcpyHostToDevice));

        material = _material_ptr.get();
        transformation = _transformation_ptr.get();

        _need_upload = false;
    }
    return _dev_ptr;
#else
    return this;
#endif
}

void Cylinder::clearGpuData()
{
#ifdef USE_CUDA
    if (_dev_ptr == nullptr)
        return;

    PX_CUDA_CHECK(hipFree(_dev_ptr));
    _dev_ptr = nullptr;
    _need_upload = true;
#endif
}

void Cylinder::setParams(Point const &center_of_bottom_face,
                         double const &radius_x, double const &radius_y,
                         double const &height)
{
    _center = center_of_bottom_face;
    _radius_x = std::abs(radius_x);
    _radius_y = std::abs(radius_y);
    _height = height;
    _abs_height = std::abs(height);

    auto top = _center.z + _height;;
    _raw_vertices[4].x = _center.x - radius_x;
    _raw_vertices[4].y = _center.y - radius_y;
    _raw_vertices[4].z = top;
    _raw_vertices[5].x = _center.x - radius_x;
    _raw_vertices[5].y = _center.y + radius_y;
    _raw_vertices[5].z = top;
    _raw_vertices[6].x = _center.x + radius_x;
    _raw_vertices[6].y = _center.y + radius_y;
    _raw_vertices[6].z = top;
    _raw_vertices[7].x = _center.x + radius_x;
    _raw_vertices[7].y = _center.y - radius_y;
    _raw_vertices[7].z = top;

    _raw_vertices[0].x = _center.x - radius_x;
    _raw_vertices[0].y = _center.y + radius_y;
    _raw_vertices[0].z = _center.z;
    _raw_vertices[1].x = _center.x + radius_x;
    _raw_vertices[1].y = _center.y + radius_y;
    _raw_vertices[1].z = _center.z;
    _raw_vertices[2].x = _center.x + radius_x;
    _raw_vertices[2].y = _center.y - radius_y;
    _raw_vertices[2].z = _center.z;
    _raw_vertices[3].x = _center.x + radius_x;
    _raw_vertices[3].y = _center.y - radius_y;
    _raw_vertices[3].z = _center.z;

    _a =  1.0 / (radius_x*radius_x);
    _b =  1.0 / (radius_y*radius_y);

    _z0 = height < 0 ? (_z1 = _center.z, top)
                     : (_z1 = top, _center.z);

#ifdef USE_CUDA
    _need_upload = true;
#endif
}

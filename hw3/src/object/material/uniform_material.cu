#include "object/material/uniform_material.hpp"

using namespace px;

BaseUniformMaterial::BaseUniformMaterial(Light const &ambient,
                                         Light const &diffuse,
                                         Light const &specular,
                                         int const &specular_exponent,
                                         Light const &transmissive,
                                         PREC const &refractive_index)
        : _ambient(ambient),
          _diffuse(diffuse),
          _specular(specular),
          _specular_exponent(specular_exponent),
          _transmissive(transmissive),
          _refractive_index(refractive_index)
{}

PX_CUDA_CALLABLE
Light BaseUniformMaterial::getAmbient(void *const &obj, PREC const &u, PREC const &v, PREC const &w)
{
    return reinterpret_cast<BaseUniformMaterial*>(obj)->_ambient;
}

PX_CUDA_CALLABLE
Light BaseUniformMaterial::getDiffuse(void *const &obj, PREC const &u, PREC const &v, PREC const &w)
{
    return reinterpret_cast<BaseUniformMaterial*>(obj)->_diffuse;
}

PX_CUDA_CALLABLE
Light BaseUniformMaterial::getSpecular(void *const &obj, PREC const &u, PREC const &v, PREC const &w)
{
    return reinterpret_cast<BaseUniformMaterial*>(obj)->_specular;
}

PX_CUDA_CALLABLE
int BaseUniformMaterial::getSpecularExp(void *const &obj, PREC const &u, PREC const &v, PREC const &w)
{
    return reinterpret_cast<BaseUniformMaterial*>(obj)->_specular_exponent;
}

PX_CUDA_CALLABLE
Light BaseUniformMaterial::getTransmissive(void *const &obj, PREC const &u, PREC const &v, PREC const &w)
{
    return reinterpret_cast<BaseUniformMaterial*>(obj)->_transmissive;
}

PX_CUDA_CALLABLE
PREC BaseUniformMaterial::getRefractiveIndex(void *const &obj, PREC const &u, PREC const &v, PREC const &w)
{
    return reinterpret_cast<BaseUniformMaterial*>(obj)->_refractive_index;
}

void BaseUniformMaterial::setAmbient(Light const &ambient)
{
    _ambient = ambient;
}
void BaseUniformMaterial::setDiffuse(Light const &diffuse)
{
    _diffuse = diffuse;
}
void BaseUniformMaterial::setSpecular(Light const &specular)
{
    _specular = specular;
}
void BaseUniformMaterial::setSpecularExp(int const &specular_exp)
{
    _specular_exponent = specular_exp;
}
void BaseUniformMaterial::setTransmissive(Light const &transmissive)
{
    _transmissive = transmissive;
}
void BaseUniformMaterial::setRefractiveIndex(PREC const &ior)
{
    _refractive_index = ior;
}

std::shared_ptr<BaseMaterial> UniformMaterial::create(Light const &ambient,
                                                Light const &diffuse,
                                                Light const &specular,
                                                int const &specular_exponent,
                                                Light const &transmissive,
                                                PREC const &refractive_index)
{
    return std::shared_ptr<BaseMaterial>(new UniformMaterial(ambient,
                                                       diffuse,
                                                       specular,
                                                       specular_exponent,
                                                       transmissive,
                                                       refractive_index));
}

UniformMaterial::UniformMaterial(Light const &ambient,
                             Light const &diffuse,
                             Light const &specular,
                             int const &specular_exponent,
                             Light const &transmissive,
                             PREC const &refractive_index)
        : BaseMaterial(),
          _obj(new BaseUniformMaterial(ambient, diffuse,
                                       specular, specular_exponent,
                                       transmissive, refractive_index)),
          _gpu_obj(nullptr),
          _need_upload(true)
{}

UniformMaterial::~UniformMaterial()
{
    delete _obj;
#ifdef USE_CUDA
    clearGpuData();
#endif
}

#ifdef USE_CUDA
__device__ fnAmbient_t __fn_ambient_uniform_material = BaseUniformMaterial::getAmbient;
__device__ fnDiffuse_t __fn_diffuse_uniform_material = BaseUniformMaterial::getDiffuse;
__device__ fnSpecular_t __fn_specular_uniform_material = BaseUniformMaterial::getSpecular;
__device__ fnSpecularExp_t __fn_specular_exp_uniform_material = BaseUniformMaterial::getSpecularExp;
__device__ fnTransmissive_t __fn_transmissive_uniform_material = BaseUniformMaterial::getTransmissive;
__device__ fnRefractiveIndex_t __fn_refractive_index_uniform_material = BaseUniformMaterial::getRefractiveIndex;
#endif

void UniformMaterial::up2Gpu()
{
#ifdef USE_CUDA
    static fnAmbient_t fn_ambient_h = nullptr;
    static fnDiffuse_t fn_diffuse_h;
    static fnSpecular_t fn_specular_h;
    static fnSpecularExp_t fn_specular_exp_h;
    static fnTransmissive_t fn_transmissive_h;
    static fnRefractiveIndex_t fn_refractive_index_h;
    
    if (_need_upload)
    {
        if (dev_ptr == nullptr)
        {
            PX_CUDA_CHECK(hipMalloc(&_gpu_obj, sizeof(BaseUniformMaterial)));
            PX_CUDA_CHECK(hipMalloc(&dev_ptr, sizeof(MaterialObj)));
        }
        if (fn_ambient_h == nullptr)
        {
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_ambient_h, HIP_SYMBOL(__fn_ambient_uniform_material), sizeof(fnAmbient_t)));
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_diffuse_h, HIP_SYMBOL(__fn_diffuse_uniform_material), sizeof(fnDiffuse_t)));
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_specular_h, HIP_SYMBOL(__fn_specular_uniform_material), sizeof(fnSpecular_t)));
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_specular_exp_h, HIP_SYMBOL(__fn_specular_exp_uniform_material), sizeof(fnSpecularExp_t)));
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_transmissive_h, HIP_SYMBOL(__fn_transmissive_uniform_material), sizeof(fnTransmissive_t)));
            PX_CUDA_CHECK(hipMemcpyFromSymbol(&fn_refractive_index_h, HIP_SYMBOL(__fn_refractive_index_uniform_material), sizeof(fnRefractiveIndex_t)));
        }
        PX_CUDA_CHECK(hipMemcpy(_gpu_obj, _obj, sizeof(BaseUniformMaterial),
                                 hipMemcpyHostToDevice));
        MaterialObj tmp(_gpu_obj,
                        fn_ambient_h, fn_diffuse_h,
                        fn_specular_h, fn_specular_exp_h,
                        fn_transmissive_h, fn_refractive_index_h);

        PX_CUDA_CHECK(hipMemcpy(dev_ptr, &tmp, sizeof(MaterialObj),
                                 hipMemcpyHostToDevice));
        _need_upload = false;
    }
#endif
}

void UniformMaterial::clearGpuData()
{
#ifdef USE_CUDA
    if (_gpu_obj != nullptr)
    {
        PX_CUDA_CHECK(hipFree(_gpu_obj));
        _gpu_obj = nullptr;
    }
    BaseMaterial::clearGpuData();
#endif
}

int UniformMaterial::specularExp(PREC const &u, PREC const &v, PREC const &w) const
{
    return BaseUniformMaterial::getSpecularExp(_obj, u, v, w);
}
PREC UniformMaterial::refractiveIndex(PREC const &u, PREC const &v, PREC const &w) const
{
    return BaseUniformMaterial::getRefractiveIndex(_obj, u, v, w);
}
Light UniformMaterial::getAmbient(PREC const &u, PREC const &v, PREC const &w) const
{
    return BaseUniformMaterial::getAmbient(_obj, u, v, w);
}
Light UniformMaterial::getDiffuse(PREC const &u, PREC const &v, PREC const &w) const
{
    return BaseUniformMaterial::getDiffuse(_obj, u, v, w);
}
Light UniformMaterial::getSpecular(PREC const &u, PREC const &v, PREC const &w) const
{
    return BaseUniformMaterial::getSpecular(_obj, u, v, w);
}
Light UniformMaterial::getTransmissive(PREC const &u, PREC const &v, PREC const &w) const
{
    return BaseUniformMaterial::getTransmissive(_obj, u, v, w);
}

void UniformMaterial::setAmbient(Light const &ambient)
{
    _obj->setAmbient(ambient);
#ifdef USE_CUDA
    _need_upload = true;
#endif
}
void UniformMaterial::setDiffuse(Light const &diffuse)
{
    _obj->setDiffuse(diffuse);
#ifdef USE_CUDA
    _need_upload = true;
#endif
}
void UniformMaterial::setSpecular(Light const &specular)
{
    _obj->setSpecular(specular);
#ifdef USE_CUDA
    _need_upload = true;
#endif
}
void UniformMaterial::setSpecularExp(int const &specular_exp)
{
    _obj->setSpecularExp(specular_exp);
#ifdef USE_CUDA
    _need_upload = true;
#endif
}
void UniformMaterial::setTransmissive(Light const &transmissive)
{
    _obj->setTransmissive(transmissive);
#ifdef USE_CUDA
    _need_upload = true;
#endif
}
void UniformMaterial::setRefractiveIndex(PREC const &ior)
{
    _obj->setRefractiveIndex(ior);
#ifdef USE_CUDA
    _need_upload = true;
#endif
}
